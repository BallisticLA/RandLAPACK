#include "hip/hip_runtime.h"
#include "RandLAPACK.hh"
#include "rl_blaspp.hh"
#include "rl_gen.hh"

#include <RandBLAS.hh>
#include <RandBLAS/test_util.hh>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <math.h>
#include <chrono>
#include <gtest/gtest.h>

// Use cuda kernels.
#ifndef USE_CUDA
#define USE_CUDA

#include "RandLAPACK/gpu_functions/rl_cuda_kernels.cuh"

using namespace std::chrono;


class TestUtil : public ::testing::Test
{
    protected:

    virtual void SetUp() {};

    virtual void TearDown() {};

    template <typename T>
    struct ColSwpTestData {
        int64_t row;
        int64_t col;
        std::vector<T> A;
        std::vector<T> A_host_buffer;
        std::vector<int64_t> J;
        T* A_device;
        T* J_device;

        ColSwpTestData(int64_t m, int64_t n) :
        A(m * n, 0.0),
        A_host_buffer(m * n, 0.0),
        J(n, 0.0)
        {
            row = m;
            col = n;
            hipMalloc(&A_device, m * n * sizeof(T));
            hipMalloc(&J_device, n * sizeof(int64_t));
        }
    };

    template <typename T>
    static void 
    test_col_swp_gpu(ColSwpTestData<T> &all_data) {

        auto m = all_data.row;
        auto n = all_data.col;
        hipStream_t strm = hipStreamPerThread;

        char host_name [] = "host";
        char device_name [] = "device";
        RandBLAS::util::print_colmaj(m, n, all_data.A.data(), host_name);

        RandLAPACK::util::col_swap(m, n, n, all_data.A.data(), m, all_data.J);
        RandLAPACK::cuda_kernels::col_swap_gpu(m, n, n, all_data.A_device, m, all_data.J_device, strm);
        hipMemcpy(all_data.A_host_buffer.data(), all_data.A_device, m * n * sizeof(T), hipMemcpyDeviceToHost);

        RandBLAS::util::print_colmaj(m, n, all_data.A_host_buffer.data(), device_name);
        RandBLAS::util::print_colmaj(m, n, all_data.A.data(), host_name);

        for(int i = 0; i < m*n; ++i)
            all_data.A[i] -= all_data.A_host_buffer[i];

        T norm_test = lapack::lange(Norm::Fro, m, n, all_data.A.data(), m);
        printf("Norm diff GPU CPU: %e\n", norm_test);
        ASSERT_NEAR(norm_test, 0.0, std::pow(std::numeric_limits<T>::epsilon(), 0.75));
    }

};

TEST_F(TestUtil, test_col_swp_gpu) {
    
    int64_t m = 5;
    int64_t n = 5;
    auto state = RandBLAS::RNGState();
    ColSwpTestData<double> all_data(m, n);

    RandLAPACK::gen::mat_gen_info<double> m_info(m, n, RandLAPACK::gen::polynomial);
    m_info.cond_num = 2025;
    m_info.rank = n;
    m_info.exponent = 2.0;
    RandLAPACK::gen::mat_gen<double, r123::Philox4x32>(m_info, all_data.A.data(), state);
    hipMemcpy(all_data.A_device, all_data.A.data(), m * n * sizeof(double), hipMemcpyHostToDevice);
    
    // Fill and randomly shuffle a vector
    std::iota(all_data.J.begin(), all_data.J.end(), 1);
    std::random_shuffle(all_data.J.begin(), all_data.J.begin() + n);
    hipMemcpy(all_data.J_device, all_data.J.data(), n * sizeof(int64_t), hipMemcpyHostToDevice);

    test_col_swp_gpu<double>(all_data);
}
#endif