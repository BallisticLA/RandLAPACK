#include "hip/hip_runtime.h"
#include "RandLAPACK.hh"
#include "rl_blaspp.hh"
#include "rl_lapackpp.hh"
#include "rl_gen.hh"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>

#include <RandBLAS.hh>
#include <fstream>
#include <gtest/gtest.h>
#include <chrono>

// Use cuda kernels.
#ifndef USE_CUDA
#define USE_CUDA
#include "RandLAPACK/drivers/rl_cqrrp_gpu.hh"

class TestCQRRP : public ::testing::TestWithParam<int64_t>
{
    protected:

    virtual void SetUp() {};

    virtual void TearDown() {};

    template <typename T>
    struct CQRRPTestData {
        int64_t row;
        int64_t col;
        int64_t rank;
        
        std::vector<T> A;
        std::vector<T> A_cpu;
        std::vector<T> A_sk;
        std::vector<T> Q;
        std::vector<T> Q_cpu;
        std::vector<T> R;
        std::vector<T> R_cpu;
        std::vector<T> R_full;
        std::vector<T> tau;
        std::vector<T> tau_cpu;
        std::vector<int64_t> J;
        std::vector<int64_t> J_cpu;
        std::vector<T> A_cpy1;
        std::vector<T> A_cpy2;
        std::vector<T> I_ref;
        
        // Buffers for the GPU data
        T* A_device;
        T* A_sk_device;
        T* tau_device;
        int64_t* J_device;

        CQRRPTestData(int64_t m, int64_t n, int64_t k, int64_t d) :
        A(m * n, 0.0),
        A_cpu(m * n, 0.0),
        A_sk(d * n, 0.0),
        Q(m * n, 0.0),
        Q_cpu(m * n, 0.0),
        R_full(m * n, 0.0),
        tau(n, 0.0),
        tau_cpu(n, 0.0),
        J(n, 0),
        J_cpu(n, 0),
        A_cpy1(m * n, 0.0),
        A_cpy2(m * n, 0.0),
        I_ref(k * k, 0.0) 
        {
            row = m;
            col = n;
            rank = k;
            hipMalloc(&A_device,    m * n * sizeof(T));
            hipMalloc(&A_sk_device, d * n * sizeof(T));
            hipMalloc(&tau_device,  n *     sizeof(T));
            hipMalloc(&J_device,    n *     sizeof(int64_t));
        }

        ~CQRRPTestData() {
            hipFree(A_device);
            hipFree(A_sk_device);
            hipFree(tau_device);
            hipFree(J_device);
        }
    };

    template <typename T>
    struct CQRRPBenchData {
        int64_t row;
        int64_t col;
        int64_t rank;
        
        std::vector<T> A;
        T* A_sk;
        // Buffers for the GPU data
        T* A_device;
        T* A_sk_device;
        T* tau_device;
        int64_t* J_device;

        CQRRPBenchData(int64_t m, int64_t n) :
        A(m * n, 0.0)
        {
            row = m;
            col = n;
            hipMalloc(&A_device,    m * n * sizeof(T));
            hipMalloc(&tau_device,  n *     sizeof(T));
            hipMalloc(&J_device,    n *     sizeof(int64_t));
        }

        ~CQRRPBenchData() {
            hipFree(A_device);
            hipFree(tau_device);
            hipFree(J_device);
        }
    };

    template <typename T, typename RNG>
    static void data_regen(
                            RandLAPACK::gen::mat_gen_info<T> m_info, 
                            CQRRPBenchData<T> &all_data, 
                            RandBLAS::RNGState<RNG> &state) {

        auto state_const = state;
        auto m = m_info.rows;
        auto n = m_info.cols;

        RandLAPACK::gen::mat_gen(m_info, all_data.A.data(), state_const);
        hipMemset(all_data.J_device, 0.0, n);
        hipMemset(all_data.tau_device, 0.0, n);
    }


    template <typename T, typename RNG>
    static void norm__sektch_and_copy_computational_helper(T &norm_A, int64_t d, CQRRPTestData<T> &all_data, RandBLAS::RNGState<RNG> &state) {

        auto m = all_data.row;
        auto n = all_data.col;
        auto state_const = state;

        // Skethcing in an sampling regime
        T* S  = ( T * ) calloc( d * m, sizeof( T ) );
        RandBLAS::DenseDist D(d, m);
        RandBLAS::fill_dense(D, S, state_const).second;
        blas::gemm(Layout::ColMajor, Op::NoTrans, Op::NoTrans, d, n, m, 1.0, S, d, all_data.A.data(), m, 0.0, all_data.A_sk.data(), d);
        free(S);
        hipMemcpy(all_data.A_sk_device, all_data.A_sk.data(), d * n * sizeof(double), hipMemcpyHostToDevice);

        hipMemcpy(all_data.A_device, all_data.A.data(), m * n * sizeof(double), hipMemcpyHostToDevice);
        lapack::lacpy(MatrixType::General, m, n, all_data.A.data(), m, all_data.A_cpu.data(), m);
        lapack::lacpy(MatrixType::General, m, n, all_data.A.data(), m, all_data.A_cpy1.data(), m);
        lapack::lacpy(MatrixType::General, m, n, all_data.A.data(), m, all_data.A_cpy2.data(), m);
        norm_A = lapack::lange(Norm::Fro, m, n, all_data.A.data(), m);
    }

    /// This routine also appears in benchmarks, but idk if it should be put into utils
    template <typename T>
    static void
    error_check(T &norm_A, CQRRPTestData<T> &all_data) {

        auto m = all_data.row;
        auto n = all_data.col;
        auto k = all_data.rank;

        RandLAPACK::util::upsize(k * k, all_data.I_ref);
        RandLAPACK::util::eye(k, k, all_data.I_ref);

        T* A_dat         = all_data.A_cpy1.data();
        T const* A_cpy_dat = all_data.A_cpy2.data();
        T const* Q_dat   = all_data.Q.data();
        T const* R_dat   = all_data.R.data();
        T* I_ref_dat     = all_data.I_ref.data();

        // Check orthogonality of Q
        // Q' * Q  - I = 0
        blas::syrk(Layout::ColMajor, Uplo::Upper, Op::Trans, k, m, 1.0, Q_dat, m, -1.0, I_ref_dat, k);
        T norm_0 = lapack::lansy(lapack::Norm::Fro, Uplo::Upper, k, I_ref_dat, k);

        // A - QR
        blas::gemm(Layout::ColMajor, Op::NoTrans, Op::NoTrans, m, n, k, 1.0, Q_dat, m, R_dat, k, -1.0, A_dat, m);
        
        // Implementing max col norm metric
        T max_col_norm = 0.0;
        T col_norm = 0.0;
        int max_idx = 0;
        for(int i = 0; i < n; ++i) {
            col_norm = blas::nrm2(m, &A_dat[m * i], 1);
            if(max_col_norm < col_norm) {
                max_col_norm = col_norm;
                max_idx = i;
            }
        }
        T col_norm_A = blas::nrm2(n, &A_cpy_dat[m * max_idx], 1);
        T norm_AQR = lapack::lange(Norm::Fro, m, n, A_dat, m);
        
        printf("REL NORM OF AP - QR:    %14e\n", norm_AQR / norm_A);
        printf("MAX COL NORM METRIC:    %14e\n", max_col_norm / col_norm_A);
        printf("FRO NORM OF (Q'Q - I):  %14e\n\n", norm_0 / std::sqrt((T) n));

        T atol = std::pow(std::numeric_limits<T>::epsilon(), 0.75);
        ASSERT_NEAR(norm_AQR / norm_A,         0.0, atol);
        ASSERT_NEAR(max_col_norm / col_norm_A, 0.0, atol);
        ASSERT_NEAR(norm_0, 0.0, atol);
    }

    /// General test for CQRRP:
    /// Computes QR factorzation, and computes A[:, J] - QR.
    template <typename T, typename RNG, typename alg_type>
    static void test_CQRRP_general(
        int64_t d, 
        T norm_A,
        CQRRPTestData<T> &all_data,
        alg_type &CQRRP_GPU) {

        auto m = all_data.row;
        auto n = all_data.col;

        CQRRP_GPU.call(m, n, all_data.A_device, m, all_data.A_sk_device, d, all_data.tau_device, all_data.J_device);

        all_data.rank = CQRRP_GPU.rank;
        printf("RANK AS RETURNED BY CQRRP GPU %4ld\n", all_data.rank);
        
        hipMemcpy(all_data.R_full.data(), all_data.A_device,   m * n * sizeof(T),   hipMemcpyDeviceToHost);
        hipMemcpy(all_data.Q.data(),      all_data.A_device,   m * n * sizeof(T),   hipMemcpyDeviceToHost);
        hipMemcpy(all_data.tau.data(),    all_data.tau_device, n * sizeof(T),       hipMemcpyDeviceToHost);
        hipMemcpy(all_data.J.data(),      all_data.J_device,   n * sizeof(int64_t), hipMemcpyDeviceToHost);

        lapack::ungqr(m, n, n, all_data.Q.data(), m, all_data.tau.data());
        RandLAPACK::util::upsize(all_data.rank * n, all_data.R);
        lapack::lacpy(MatrixType::Upper, all_data.rank, n, all_data.R_full.data(), m, all_data.R.data(), all_data.rank);

        RandLAPACK::util::col_swap(m, n, n, all_data.A_cpy1.data(), m, all_data.J);
        RandLAPACK::util::col_swap(m, n, n, all_data.A_cpy2.data(), m, all_data.J);

        error_check(norm_A, all_data);
    }

    template <typename T, typename RNG, typename alg_gpu, typename alg_cpu>
    static void test_CQRRP_compare_with_CPU(
        int64_t d, 
        T norm_A,
        CQRRPTestData<T> &all_data,
        alg_gpu &CQRRP_GPU,
        alg_cpu &CQRRP_CPU,
        RandBLAS::RNGState<RNG> &state) {

        auto m = all_data.row;
        auto n = all_data.col;

        CQRRP_GPU.call(m, n, all_data.A_device, m, all_data.A_sk_device, d, all_data.tau_device, all_data.J_device);
        CQRRP_CPU.call(m, n, all_data.A_cpu.data(), m, (T) (d / CQRRP_CPU.block_size) , all_data.tau_cpu.data(), all_data.J_cpu.data(), state);
        
        hipMemcpy(all_data.R_full.data(), all_data.A_device,   m * n * sizeof(T),   hipMemcpyDeviceToHost);
        hipMemcpy(all_data.tau.data(),    all_data.tau_device, n * sizeof(T),       hipMemcpyDeviceToHost);
        hipMemcpy(all_data.J.data(),      all_data.J_device,   n * sizeof(int64_t), hipMemcpyDeviceToHost);

        for(int i = 0; i < n; ++i) {
            all_data.tau[i] -= all_data.tau_cpu[i];
            all_data.J[i] -= all_data.J_cpu[i];

            for(int j = 0; j <= i; ++j) {
                all_data.A_cpu[i * m + j] -= all_data.R_full[i * m + j];
            }
        }
        RandLAPACK::util::get_U(n, n, all_data.A_cpu.data(), m);

        T col_nrm_J   = blas::nrm2(n, all_data.J.data(), 1);
        T col_nrm_tau = blas::nrm2(n, all_data.tau.data(), 1);
        T norm_R_diff = lapack::lange(Norm::Fro, n, n, all_data.A_cpu.data(), m);

        T atol1 = std::pow(std::numeric_limits<T>::epsilon(), 0.75);
        T atol2 = std::pow(std::numeric_limits<T>::epsilon(), 0.60);
        ASSERT_NEAR(col_nrm_J,   0.0, atol1);
        ASSERT_NEAR(col_nrm_tau, 0.0, atol1);
        ASSERT_NEAR(norm_R_diff, 0.0, atol2);
    }

    template <typename T, typename RNG>
    static std::vector<long> bench_CQRRP(
        RandLAPACK::gen::mat_gen_info<T> m_info,
        int64_t d_factor, 
        T tol,
        int64_t block_size,
        CQRRPBenchData<T> &all_data,
        RandBLAS::RNGState<RNG> state) {

        auto m = all_data.row;
        auto n = all_data.col;
        auto state_const = state;
        auto d = d_factor * block_size;

        // Skethcing in an sampling regime
        hipMalloc(&all_data.A_sk_device, d * n * sizeof(T));
        all_data.A_sk  = ( T * ) calloc( d * n, sizeof( T ) );
        T* S           = ( T * ) calloc( d * m, sizeof( T ) );
        RandBLAS::DenseDist D(d, m);
        RandBLAS::fill_dense(D, S, state_const).second;
        blas::gemm(Layout::ColMajor, Op::NoTrans, Op::NoTrans, d, n, m, 1.0, S, d, all_data.A.data(), m, 0.0, all_data.A_sk, d);
        free(S);
        hipMemcpy(all_data.A_sk_device, all_data.A_sk, d * n * sizeof(double), hipMemcpyHostToDevice);
	
        RandLAPACK::CQRRP_blocked_GPU<double, r123::Philox4x32> CQRRP_GPU(true, tol, block_size);
	    auto start = std::chrono::steady_clock::now();
        CQRRP_GPU.call(m, n, all_data.A_device, m, all_data.A_sk_device, d, all_data.tau_device, all_data.J_device);
	    auto stop = std::chrono::steady_clock::now();
	    auto diff = std::chrono::duration_cast<std::chrono::milliseconds>(stop - start).count();
        auto rank = CQRRP_GPU.rank;
        //printf("RANK AS RETURNED BY CQRRP GPU %4ld\n", rank);
	    printf("  BLOCK SIZE = %ld TIME (MS) = %ld\n", block_size, diff);

        data_regen(m_info, all_data, state);

        hipFree(all_data.A_sk_device);
        free(all_data.A_sk);

        return CQRRP_GPU.times;
    }

};

// Note: If Subprocess killed exception -> reload vscode
TEST_F(TestCQRRP, CQRRP_GPU_070824) {
    int64_t m = 5000;//5000;
    int64_t n = 2800;//2000;
    int64_t k = 2800;
    double d_factor = 1;//1.0;
    int64_t b_sz = 900;//500;
    int64_t d = d_factor * b_sz;
    double norm_A = 0;
    double tol = std::pow(std::numeric_limits<double>::epsilon(), 0.85);
    auto state = RandBLAS::RNGState();

    CQRRPTestData<double> all_data(m, n, k, d);
    RandLAPACK::CQRRP_blocked_GPU<double, r123::Philox4x32> CQRRP_blocked_GPU(true, tol, b_sz);

    RandLAPACK::gen::mat_gen_info<double> m_info(m, n, RandLAPACK::gen::gaussian);
    //RandLAPACK::gen::mat_gen_info<double> m_info(m, n, RandLAPACK::gen::polynomial);
    //m_info.cond_num = 2;
    //m_info.rank = k;
    //m_info.exponent = 2.0;
    RandLAPACK::gen::mat_gen<double, r123::Philox4x32>(m_info, all_data.A.data(), state);

    norm__sektch_and_copy_computational_helper<double, r123::Philox4x32>(norm_A, d, all_data, state);
#if !defined(__APPLE__)
    test_CQRRP_general<double, RandLAPACK::CQRRP_blocked_GPU<double, r123::Philox4x32>>(d, norm_A, all_data, CQRRP_blocked_GPU);
#endif
}

// Note: If Subprocess killed exception -> reload vscode
TEST_F(TestCQRRP, CQRRP_GPU_vectors) {
    int64_t m = 5000;//5000;
    int64_t n = 2800;//2000;
    int64_t k = 2800;
    double d_factor = 1;//1.0;
    int64_t b_sz = 900;//500;
    int64_t d = d_factor * b_sz;
    double norm_A = 0;
    double tol = std::pow(std::numeric_limits<double>::epsilon(), 0.85);
    auto state = RandBLAS::RNGState();

    CQRRPTestData<double> all_data(m, n, k, d);
    RandLAPACK::CQRRP_blocked_GPU<double, r123::Philox4x32> CQRRP_blocked_GPU(false, tol, b_sz);
    RandLAPACK::CQRRP_blocked<double, r123::Philox4x32> CQRRP_blocked_CPU(false, tol, b_sz);
    CQRRP_blocked_CPU.nnz = 2;
    CQRRP_blocked_CPU.num_threads = 4;
    CQRRP_blocked_CPU.use_gaussian = true;

    RandLAPACK::gen::mat_gen_info<double> m_info(m, n, RandLAPACK::gen::gaussian);
    RandLAPACK::gen::mat_gen<double, r123::Philox4x32>(m_info, all_data.A.data(), state);

    norm__sektch_and_copy_computational_helper<double, r123::Philox4x32>(norm_A, d, all_data, state);
#if !defined(__APPLE__)
    test_CQRRP_compare_with_CPU(d, norm_A, all_data, CQRRP_blocked_GPU, CQRRP_blocked_CPU, state);
#endif
}

TEST_P(TestCQRRP, CQRRP_GPU_benchmark_16k) {
    int64_t m = std::pow(2, 14);
    int64_t n = std::pow(2, 14);
    double d_factor = 1.25;
    int64_t b_sz    = GetParam();
    double tol = std::pow(std::numeric_limits<double>::epsilon(), 0.85);
    auto state = RandBLAS::RNGState();

    CQRRPBenchData<double> all_data(m, n);
    RandLAPACK::gen::mat_gen_info<double> m_info(m, n, RandLAPACK::gen::gaussian);
    RandLAPACK::gen::mat_gen<double, r123::Philox4x32>(m_info, all_data.A.data(), state);
    hipMemcpy(all_data.A_device, all_data.A.data(), m * n * sizeof(double), hipMemcpyHostToDevice);


    std::fstream file("ICQRRP_gpu_runtime_breakdown_"  + std::to_string(m)
                                    + "_d_factor_"     + std::to_string(d_factor)
                                    + ".dat", std::fstream::app);
    auto res = bench_CQRRP(m_info, d_factor, tol, b_sz, all_data, state);
    file << res[0]  << ",  " << res[1]  << ",  " << res[2] << ",  " << res[3] << ",  " << res[4] << ",  " << res[5] << ",  " << res[6] << ",  " << res[7] << ",  " << res[8] << ",  " << res[9] << ",  " << res[10] << ",  " << res[11] << res[12] << ",  " << res[13] << ",\n";
}


INSTANTIATE_TEST_SUITE_P(
    CQRRP_GPU_16k_benchmarks,
    TestCQRRP,
    ::testing::Values(32)//, 40, 48, 56, 64, 72, 80, 88, 96, 104, 112, 120, 128, 136, 144, 152, 160, 168, 176, 184, 192)
);

#endif
