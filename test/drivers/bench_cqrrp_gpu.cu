#include "hip/hip_runtime.h"
#include "RandLAPACK.hh"
#include "rl_blaspp.hh"
#include "rl_lapackpp.hh"
#include "rl_gen.hh"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>

#include <RandBLAS.hh>
#include <fstream>
#include <gtest/gtest.h>
#include <chrono>

// Use cuda kernels.
#ifndef USE_CUDA
#define USE_CUDA
#include "RandLAPACK/drivers/rl_cqrrp_gpu.hh"

class BenchCQRRP : public ::testing::TestWithParam<int64_t>
{
    protected:

    virtual void SetUp() {};

    virtual void TearDown() {};

    template <typename T>
    struct CQRRPBenchData {
        int64_t row;
        int64_t col;
        int64_t rank;
        
        std::vector<T> A;
        T* A_sk;
        // Buffers for the GPU data
        T* A_device;
        T* A_sk_device;
        T* tau_device;
        int64_t* J_device;

        T* R_device;
        T* D_device;

        CQRRPBenchData(int64_t m, int64_t n) :
        A(m * n, 0.0)
        {
            row = m;
            col = n;
            hipMalloc(&A_device,    m * n * sizeof(T));
            hipMalloc(&tau_device,  n *     sizeof(T));
            hipMalloc(&J_device,    n *     sizeof(int64_t));
            hipMalloc(&R_device,    n * n * sizeof(T));
            hipMalloc(&D_device,    n *     sizeof(T));
        }

        ~CQRRPBenchData() {
            hipFree(A_device);
            hipFree(tau_device);
            hipFree(J_device);
            hipFree(R_device);
            hipFree(D_device);
        }
    };

    template <typename T, typename RNG>
    static void data_regen(
                            RandLAPACK::gen::mat_gen_info<T> m_info, 
                            CQRRPBenchData<T> &all_data, 
                            RandBLAS::RNGState<RNG> &state) {

        auto state_const = state;
        auto m = m_info.rows;
        auto n = m_info.cols;

        RandLAPACK::gen::mat_gen(m_info, all_data.A.data(), state_const);
        hipMemset(all_data.J_device, 0.0, n);
        hipMemset(all_data.tau_device, 0.0, n);
    }

    template <typename T, typename RNG>
    static void bench_CQRRP(
        bool profile_runtime,
        bool run_qrf,
	bool cqrrp_use_qrf,
        RandLAPACK::gen::mat_gen_info<T> m_info,
        T tol,
        int64_t block_size,
        CQRRPBenchData<T> &all_data,
        RandBLAS::RNGState<RNG> state,
        std::string output_filename_breakdown,
        std::string output_filename_speed) {

	T d_factor = 1.0;
        auto m = all_data.row;
        auto n = all_data.col;
        auto state_const = state;
        int64_t d = d_factor * block_size;

        // Skethcing in an sampling regime
        hipMalloc(&all_data.A_sk_device, d * n * sizeof(T));
        all_data.A_sk  = ( T * ) calloc( d * n, sizeof( T ) );
        T* S           = ( T * ) calloc( d * m, sizeof( T ) );
        RandBLAS::DenseDist D(d, m);
        RandBLAS::fill_dense(D, S, state_const).second;
        blas::gemm(Layout::ColMajor, Op::NoTrans, Op::NoTrans, d, n, m, 1.0, S, d, all_data.A.data(), m, 0.0, all_data.A_sk, d);
        free(S);
        hipMemcpy(all_data.A_sk_device, all_data.A_sk, d * n * sizeof(double), hipMemcpyHostToDevice);
	
        RandLAPACK::CQRRP_blocked_GPU<double, r123::Philox4x32> CQRRP_GPU(profile_runtime, tol, block_size);
        CQRRP_GPU.use_qrf = cqrrp_use_qrf;
	    auto start = std::chrono::steady_clock::now();
        CQRRP_GPU.call(m, n, all_data.A_device, m, all_data.A_sk_device, d, all_data.tau_device, all_data.J_device);
	    auto stop = std::chrono::steady_clock::now();
	    auto diff_icqrrp = std::chrono::duration_cast<std::chrono::milliseconds>(stop - start).count();
        auto rank = CQRRP_GPU.rank;
        data_regen(m_info, all_data, state);
        hipFree(all_data.A_sk_device);
        free(all_data.A_sk);

        if(profile_runtime) {
            std::ofstream file(output_filename_breakdown, std::ios::app);
            std::copy(CQRRP_GPU.times.data(), CQRRP_GPU.times.data() + 17, std::ostream_iterator<T>(file, ", "));
            file << "\n";
        } 
        long diff_qrf = 0;
        if (run_qrf) {
            lapack::Queue lapack_queue(0);
            using lapack::device_info_int;
            device_info_int* d_info = blas::device_malloc< device_info_int >( 1, lapack_queue );
            char* d_work_geqrf;
            char* h_work_geqrf;
            size_t d_size_geqrf, h_size_geqrf;
            
            auto start_qrf = std::chrono::steady_clock::now();
            lapack::geqrf_work_size_bytes(m, n, all_data.A_device, m, &d_size_geqrf, &h_size_geqrf, lapack_queue);
            d_work_geqrf = blas::device_malloc< char >( d_size_geqrf, lapack_queue );
            std::vector<char> h_work_geqrf_vector( h_size_geqrf );
            h_work_geqrf = h_work_geqrf_vector.data();
            lapack::geqrf(m, n, all_data.A_device, m, all_data.tau_device, d_work_geqrf, d_size_geqrf, h_work_geqrf, h_size_geqrf, d_info, lapack_queue);
            lapack_queue.sync();
            auto stop_qrf  = std::chrono::steady_clock::now();
	        diff_qrf  = std::chrono::duration_cast<std::chrono::milliseconds>(stop_qrf  - start_qrf).count();
            printf(" QRF TIME (MS) = %ld\n", diff_qrf);
        }

	    printf("  BLOCK SIZE = %ld TIME (MS) = %ld\n", block_size, diff_icqrrp);
        std::ofstream file(output_filename_speed, std::ios::app);
        file << m << "  " << n << "  " << block_size << "  " << diff_icqrrp << "  " << diff_qrf << "\n";
    }

    template <typename T, typename RNG>
    static void bench_CholQR(
        RandLAPACK::gen::mat_gen_info<T> m_info,
        int64_t numcols,
        CQRRPBenchData<T> &all_data,
        RandBLAS::RNGState<RNG> state,
        std::string output_filename) {

        auto m = all_data.row;
        auto n = numcols;
        auto state_const = state;

        // Initialize GPU stuff
        lapack::Queue lapack_queue(0);
        hipStream_t strm = lapack_queue.stream();
        using lapack::device_info_int;
        device_info_int* d_info = blas::device_malloc< device_info_int >( 1, lapack_queue );
        char* d_work_geqrf;
        char* h_work_geqrf;
        size_t d_size_geqrf, h_size_geqrf;

        // CholQR part
        auto start_cholqr = std::chrono::steady_clock::now();
        blas::syrk(Layout::ColMajor, Uplo::Upper, Op::Trans, n, m, (T) 1.0, all_data.A_device, m, (T) 0.0, all_data.R_device, n, lapack_queue);
        lapack::potrf(Uplo::Upper,  n, all_data.R_device, n, d_info, lapack_queue);
        blas::trsm(Layout::ColMajor, Side::Right, Uplo::Upper, Op::NoTrans, Diag::NonUnit, m, n, (T) 1.0, all_data.R_device, n, all_data.A_device, m, lapack_queue);
        lapack_queue.sync();
        auto stop_cholqr  = std::chrono::steady_clock::now();
        auto diff_cholqr  = std::chrono::duration_cast<std::chrono::milliseconds>(stop_cholqr  - start_cholqr).count();
        
        auto start_orhr_col = std::chrono::steady_clock::now();
        // ORHR_COL part
        RandLAPACK::cuda_kernels::orhr_col_gpu(strm, m, n, all_data.A_device, m, all_data.tau_device, all_data.D_device);  
        RandLAPACK::cuda_kernels::R_cholqr_signs_gpu(strm, n, n, all_data.R_device, all_data.D_device);
        hipStreamSynchronize(strm);
        auto stop_orhr_col  = std::chrono::steady_clock::now();
        auto diff_orhr_col  = std::chrono::duration_cast<std::chrono::milliseconds>(stop_orhr_col  - start_orhr_col).count();

        // Mandatory data re-generation
        data_regen(m_info, all_data, state);

        // QRF part
        auto start_qrf = std::chrono::steady_clock::now();
        lapack::geqrf_work_size_bytes(m, n, all_data.A_device, m, &d_size_geqrf, &h_size_geqrf, lapack_queue);
        d_work_geqrf = blas::device_malloc< char >( d_size_geqrf, lapack_queue );
        std::vector<char> h_work_geqrf_vector( h_size_geqrf );
        h_work_geqrf = h_work_geqrf_vector.data();
        lapack::geqrf(m, n, all_data.A_device, m, all_data.tau_device, d_work_geqrf, d_size_geqrf, h_work_geqrf, h_size_geqrf, d_info, lapack_queue);
        lapack_queue.sync();
        auto stop_qrf  = std::chrono::steady_clock::now();
        auto diff_qrf  = std::chrono::duration_cast<std::chrono::milliseconds>(stop_qrf  - start_qrf).count();
        printf(" CholQR TIME (MS)   = %ld\n", diff_cholqr);
        printf(" ORHR_COL TIME (MS) = %ld\n", diff_orhr_col);
        printf(" QRF TIME (MS)      = %ld\n", diff_qrf);

        std::ofstream file(output_filename, std::ios::app);
        file << m << "  " << n << "  " << diff_cholqr << "  " << diff_orhr_col << "  " << diff_qrf << "\n";
    }

};
TEST_P(BenchCQRRP, CQRRP_GPU_benchmark_16k) {
    int64_t m            = std::pow(2, 14);
    int64_t n            = std::pow(2, 14);
    int64_t b_sz         = GetParam();
    double tol           = std::pow(std::numeric_limits<double>::epsilon(), 0.85);
    auto state           = RandBLAS::RNGState();
    bool profile_runtime = true;
    bool run_qrf         = false;
    bool cqrrp_uses_qrf  = true;
    if(b_sz == 120 || b_sz == 128) {
        run_qrf = true;
    }

    CQRRPBenchData<double> all_data(m, n);
    RandLAPACK::gen::mat_gen_info<double> m_info(m, n, RandLAPACK::gen::gaussian);
    RandLAPACK::gen::mat_gen<double, r123::Philox4x32>(m_info, all_data.A.data(), state);
    hipMemcpy(all_data.A_device, all_data.A.data(), m * n * sizeof(double), hipMemcpyHostToDevice);


    std::string file1 = "ICQRRP_GPU_runtime_breakdown_innerQRF_"              
	                                              + std::to_string(cqrrp_uses_qrf) 
	                            +  "_rows_"       + std::to_string(m)
                                    +  "_cols_"       + std::to_string(n)
                                    +  "_d_factor_1.0.dat";

    std::string file2 = "ICQRRP_GPU_speed_innerQRF_"
                                                      + std::to_string(cqrrp_uses_qrf)
	                            + "_rows_"        + std::to_string(m)
                                    + "_cols_"        + std::to_string(n)
                                    + "_d_factor_1.0.dat";

    bench_CQRRP(profile_runtime, run_qrf, cqrrp_uses_qrf,  m_info, tol, b_sz, all_data, state, file1, file2);
}

INSTANTIATE_TEST_SUITE_P(
    CQRRP_GPU_16k_benchmarks,
    BenchCQRRP,
    ::testing::Values(32, 64, 96, 128, 160, 192, 224, 256, 288, 320, 352, 384, 416, 448, 480, 512)
);

TEST_F(BenchCQRRP, Bench_CholQR) {
    int64_t m       = std::pow(2, 14);
    int64_t n_start = 288;
    int64_t n_stop  = std::pow(2, 14);
    auto state      = RandBLAS::RNGState();

    CQRRPBenchData<double> all_data(m, n_stop);
    RandLAPACK::gen::mat_gen_info<double> m_info(m, n_stop, RandLAPACK::gen::gaussian);
    RandLAPACK::gen::mat_gen<double, r123::Philox4x32>(m_info, all_data.A.data(), state);
    hipMemcpy(all_data.A_device, all_data.A.data(), m * n_stop * sizeof(double), hipMemcpyHostToDevice);


    std::string file = "CholQR_GPU_speed_rows_"      + std::to_string(m)
                                    + "_cols_start_" + std::to_string(n_start)
                                    + "_cols_stop_"  + std::to_string(n_stop)
                                    + ".dat";

    for(int i = n_start; i <= n_stop; i += n_start)
        bench_CholQR(m_info, i, all_data, state, file);
}
#endif
