#include "hip/hip_runtime.h"
#include "RandLAPACK.hh"
#include "rl_blaspp.hh"
#include "rl_lapackpp.hh"
#include "rl_gen.hh"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>

#include <RandBLAS.hh>
#include <fstream>
#include <gtest/gtest.h>
#include <chrono>

// Use cuda kernels.
#ifndef USE_CUDA
#define USE_CUDA
#include "RandLAPACK/drivers/rl_cqrrp_gpu.hh"

class BenchCQRRP : public ::testing::TestWithParam<int64_t>
{
    protected:

    virtual void SetUp() {};

    virtual void TearDown() {};

    template <typename T>
    struct CQRRPBenchData {
        int64_t row;
        int64_t col;
        int64_t rank;
        
        std::vector<T> A;
        T* A_sk;
        // Buffers for the GPU data
        T* A_device;
        T* A_sk_device;
        T* tau_device;
        int64_t* J_device;

        T* R_device;
        T* D_device;

        CQRRPBenchData(int64_t m, int64_t n) :
        A(m * n, 0.0)
        {
            row = m;
            col = n;
            hipMalloc(&A_device,    m * n * sizeof(T));
            hipMalloc(&tau_device,  n *     sizeof(T));
            hipMalloc(&J_device,    n *     sizeof(int64_t));
            hipMalloc(&R_device,    n * n * sizeof(T));
            hipMalloc(&D_device,    n *     sizeof(T));
        }

        ~CQRRPBenchData() {
            hipFree(A_device);
            hipFree(tau_device);
            hipFree(J_device);
            hipFree(R_device);
            hipFree(D_device);
        }
    };

    template <typename T, typename RNG>
    static void data_regen(
                            RandLAPACK::gen::mat_gen_info<T> m_info, 
                            CQRRPBenchData<T> &all_data, 
                            RandBLAS::RNGState<RNG> &state) {

        auto state_const = state;
        auto m = m_info.rows;
        auto n = m_info.cols;

        RandLAPACK::gen::mat_gen(m_info, all_data.A.data(), state_const);
        hipMemset(all_data.J_device, 0.0, n);
        hipMemset(all_data.tau_device, 0.0, n);
    }

    template <typename T, typename RNG>
    static void bench_CQRRP(
        bool profile_runtime,
        bool run_qrf,
        RandLAPACK::gen::mat_gen_info<T> m_info,
        T tol,
        int64_t block_size,
        CQRRPBenchData<T> &all_data,
        RandBLAS::RNGState<RNG> state,
        std::string output_filename_breakdown_QRF,
        std::string output_filename_breakdown_CholQR,
        std::string output_filename_speed) {

	    T d_factor = 1.0;
        auto m = all_data.row;
        auto n = all_data.col;
        auto state_const = state;
        int64_t d = d_factor * block_size;

        // ICQRRP with QRF
        // Skethcing in an sampling regime
        hipMalloc(&all_data.A_sk_device, d * n * sizeof(T));
        all_data.A_sk = ( T * ) calloc( d * n, sizeof( T ) );
        T* S          = ( T * ) calloc( d * m, sizeof( T ) );
        RandBLAS::DenseDist D(d, m);
        RandBLAS::fill_dense(D, S, state_const).second;
        blas::gemm(Layout::ColMajor, Op::NoTrans, Op::NoTrans, d, n, m, 1.0, S, d, all_data.A.data(), m, 0.0, all_data.A_sk, d);
        hipMemcpy(all_data.A_sk_device, all_data.A_sk, d * n * sizeof(double), hipMemcpyHostToDevice);
        RandLAPACK::CQRRP_blocked_GPU<double, r123::Philox4x32> CQRRP_GPU_QRF(profile_runtime, tol, block_size);
        CQRRP_GPU_QRF.use_qrf = true;
	    auto start_icqrrp_qrf = std::chrono::steady_clock::now();
        CQRRP_GPU_QRF.call(m, n, all_data.A_device, m, all_data.A_sk_device, d, all_data.tau_device, all_data.J_device);
	    auto stop_icqrrp_qrf = std::chrono::steady_clock::now();
	    auto diff_icqrrp_qrf = std::chrono::duration_cast<std::chrono::milliseconds>(stop_icqrrp_qrf - start_icqrrp_qrf).count();
        data_regen(m_info, all_data, state);
        hipFree(all_data.A_sk_device);
        free(all_data.A_sk);

        if(profile_runtime) {
            std::ofstream file(output_filename_breakdown_QRF, std::ios::app);
            std::copy(CQRRP_GPU_QRF.times.data(), CQRRP_GPU_QRF.times.data() + 17, std::ostream_iterator<T>(file, ", "));
            file << "\n";
        } 

        // ICQRRP with CholQR
        // Skethcing in an sampling regime
        hipMalloc(&all_data.A_sk_device, d * n * sizeof(T));
        all_data.A_sk = ( T * ) calloc( d * n, sizeof( T ) );
        blas::gemm(Layout::ColMajor, Op::NoTrans, Op::NoTrans, d, n, m, 1.0, S, d, all_data.A.data(), m, 0.0, all_data.A_sk, d);
        free(S);
        hipMemcpy(all_data.A_sk_device, all_data.A_sk, d * n * sizeof(double), hipMemcpyHostToDevice);
        RandLAPACK::CQRRP_blocked_GPU<double, r123::Philox4x32> CQRRP_GPU_CholQR(profile_runtime, tol, block_size);
        CQRRP_GPU_CholQR.use_qrf = false;
	    auto start_icqrrp_cholqr = std::chrono::steady_clock::now();
        CQRRP_GPU_CholQR.call(m, n, all_data.A_device, m, all_data.A_sk_device, d, all_data.tau_device, all_data.J_device);
	    auto stop_icqrrp_cholqr = std::chrono::steady_clock::now();
	    auto diff_icqrrp_cholqr = std::chrono::duration_cast<std::chrono::milliseconds>(stop_icqrrp_cholqr - start_icqrrp_cholqr).count();
        data_regen(m_info, all_data, state);
        hipFree(all_data.A_sk_device);
        free(all_data.A_sk);

        if(profile_runtime) {
            std::ofstream file(output_filename_breakdown_CholQR, std::ios::app);
            std::copy(CQRRP_GPU_CholQR.times.data(), CQRRP_GPU_CholQR.times.data() + 17, std::ostream_iterator<T>(file, ", "));
            file << "\n";
        } 

        // Optional QRF
        long diff_qrf = 0;
        if (run_qrf) {
            lapack::Queue lapack_queue(0);
            using lapack::device_info_int;
            device_info_int* d_info = blas::device_malloc< device_info_int >( 1, lapack_queue );
            char* d_work_geqrf;
            char* h_work_geqrf;
            size_t d_size_geqrf, h_size_geqrf;
            
            auto start_qrf = std::chrono::steady_clock::now();
            lapack::geqrf_work_size_bytes(m, n, all_data.A_device, m, &d_size_geqrf, &h_size_geqrf, lapack_queue);
            d_work_geqrf = blas::device_malloc< char >( d_size_geqrf, lapack_queue );
            std::vector<char> h_work_geqrf_vector( h_size_geqrf );
            h_work_geqrf = h_work_geqrf_vector.data();
            lapack::geqrf(m, n, all_data.A_device, m, all_data.tau_device, d_work_geqrf, d_size_geqrf, h_work_geqrf, h_size_geqrf, d_info, lapack_queue);
            lapack_queue.sync();
            auto stop_qrf  = std::chrono::steady_clock::now();
	        diff_qrf  = std::chrono::duration_cast<std::chrono::milliseconds>(stop_qrf  - start_qrf).count();
            printf(" QRF TIME (MS) = %ld\n", diff_qrf);
        }

	    printf("  BLOCK SIZE = %ld ICQRRP+QRF TIME (MS) = %ld ICQRRP+CholQR TIME (MS) = %ld\n", block_size, diff_icqrrp_qrf, diff_icqrrp_cholqr);
        std::ofstream file(output_filename_speed, std::ios::app);
        file << m << "  " << n << "  " << block_size << "  " << diff_icqrrp_qrf << "  " << diff_icqrrp_cholqr << "  " << diff_qrf << "\n";
    }

    // Not using this right now. But there's no harm in keeping it around.
    template <typename T, typename RNG>
    static void bench_CholQR(
        RandLAPACK::gen::mat_gen_info<T> m_info,
        int64_t numrows,
        CQRRPBenchData<T> &all_data,
        RandBLAS::RNGState<RNG> state,
        std::string output_filename) {

        auto m = numrows;
        auto n = all_data.col;
        auto state_const = state;

        // Initialize GPU stuff
        lapack::Queue lapack_queue(0);
        hipStream_t strm = lapack_queue.stream();
        using lapack::device_info_int;
        device_info_int* d_info = blas::device_malloc< device_info_int >( 1, lapack_queue );
        char* d_work_geqrf;
        char* h_work_geqrf;
        size_t d_size_geqrf, h_size_geqrf;

        // CholQR part
        auto start_cholqr = std::chrono::steady_clock::now();
        blas::syrk(Layout::ColMajor, Uplo::Upper, Op::Trans, n, m, (T) 1.0, all_data.A_device, m, (T) 0.0, all_data.R_device, n, lapack_queue);
        lapack::potrf(Uplo::Upper,  n, all_data.R_device, n, d_info, lapack_queue);
        blas::trsm(Layout::ColMajor, Side::Right, Uplo::Upper, Op::NoTrans, Diag::NonUnit, m, n, (T) 1.0, all_data.R_device, n, all_data.A_device, m, lapack_queue);
        lapack_queue.sync();
        auto stop_cholqr  = std::chrono::steady_clock::now();
        auto diff_cholqr  = std::chrono::duration_cast<std::chrono::milliseconds>(stop_cholqr  - start_cholqr).count();
        
        auto start_orhr_col = std::chrono::steady_clock::now();
        // ORHR_COL part
        RandLAPACK::cuda_kernels::orhr_col_gpu(strm, m, n, all_data.A_device, m, all_data.tau_device, all_data.D_device);  
        RandLAPACK::cuda_kernels::R_cholqr_signs_gpu(strm, n, n, all_data.R_device, all_data.D_device);
        hipStreamSynchronize(strm);
        auto stop_orhr_col  = std::chrono::steady_clock::now();
        auto diff_orhr_col  = std::chrono::duration_cast<std::chrono::milliseconds>(stop_orhr_col  - start_orhr_col).count();

        // Mandatory data re-generation
        data_regen(m_info, all_data, state);

        // QRF part
        auto start_qrf = std::chrono::steady_clock::now();
        lapack::geqrf_work_size_bytes(m, n, all_data.A_device, m, &d_size_geqrf, &h_size_geqrf, lapack_queue);
        d_work_geqrf = blas::device_malloc< char >( d_size_geqrf, lapack_queue );
        std::vector<char> h_work_geqrf_vector( h_size_geqrf );
        h_work_geqrf = h_work_geqrf_vector.data();
        lapack::geqrf(m, n, all_data.A_device, m, all_data.tau_device, d_work_geqrf, d_size_geqrf, h_work_geqrf, h_size_geqrf, d_info, lapack_queue);
        lapack_queue.sync();
        auto stop_qrf  = std::chrono::steady_clock::now();
        auto diff_qrf  = std::chrono::duration_cast<std::chrono::milliseconds>(stop_qrf  - start_qrf).count();
        printf(" CholQR TIME (MS)   = %ld\n", diff_cholqr);
        printf(" ORHR_COL TIME (MS) = %ld\n", diff_orhr_col);
        printf(" QRF TIME (MS)      = %ld\n", diff_qrf);

        std::ofstream file(output_filename, std::ios::app);
        file << m << "  " << n << "  " << diff_cholqr << "  " << diff_orhr_col << "  " << diff_qrf << "\n";
    }

};

TEST_P(BenchCQRRP, GPU_fixed_blocksize) {
    int64_t m            = std::pow(2, 15);
    int64_t n            = std::pow(2, 15);
    int64_t b_sz         = GetParam();
    double tol           = std::pow(std::numeric_limits<double>::epsilon(), 0.85);
    auto state           = RandBLAS::RNGState();
    bool profile_runtime = true;
    bool run_qrf         = false;
    if(b_sz == 128) {
        run_qrf = true;
    }

    CQRRPBenchData<double> all_data(m, n);
    RandLAPACK::gen::mat_gen_info<double> m_info(m, n, RandLAPACK::gen::gaussian);
    RandLAPACK::gen::mat_gen<double, r123::Philox4x32>(m_info, all_data.A.data(), state);
    hipMemcpy(all_data.A_device, all_data.A.data(), m * n * sizeof(double), hipMemcpyHostToDevice);


    std::string file1 = "ICQRRP_GPU_runtime_breakdown_innerQRF_1_rows_"       
                                                      + std::to_string(m)
                                    +  "_cols_"       + std::to_string(n)
                                    +  "_d_factor_1.0.dat";

    std::string file2 = "ICQRRP_GPU_runtime_breakdown_innerQRF_0_rows_"       
                                                    + std::to_string(m)
                                +  "_cols_"       + std::to_string(n)
                                +  "_d_factor_1.0.dat";

    std::string file3 = "ICQRRP_GPU_speed_rows_"      
                                                      + std::to_string(m)
                                    + "_cols_"        + std::to_string(n)
                                    + "_d_factor_1.0.dat";

    bench_CQRRP(profile_runtime, run_qrf, m_info, tol, b_sz, all_data, state, file1, file2, file3);
}

INSTANTIATE_TEST_SUITE_P(
    CQRRP_GPU_benchmarks,
    BenchCQRRP,
    ::testing::Values(32, 64, 96, 128, 160, 192, 224, 256, 288, 320, 352, 384, 416, 448, 480, 512, 640, 768, 896, 1024, 1152, 1280, 1408, 1536, 1664, 1792, 1920, 2048)
);
#endif
