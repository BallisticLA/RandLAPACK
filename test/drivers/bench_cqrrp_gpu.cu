#include "hip/hip_runtime.h"
#include "RandLAPACK.hh"
#include "rl_blaspp.hh"
#include "rl_lapackpp.hh"
#include "rl_gen.hh"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>

#include <RandBLAS.hh>
#include <fstream>
#include <gtest/gtest.h>
#include <chrono>

// Use cuda kernels.
#ifndef USE_CUDA
#define USE_CUDA
#include "RandLAPACK/drivers/rl_cqrrp_gpu.hh"

class BenchCQRRP : public ::testing::TestWithParam<int64_t>
{
    protected:

    virtual void SetUp() {};

    virtual void TearDown() {};

    template <typename T>
    struct CQRRPBenchData {
        int64_t row;
        int64_t col;
        int64_t rank;
        
        std::vector<T> A;
        T* A_sk;
        // Buffers for the GPU data
        T* A_device;
        T* A_sk_device;
        T* tau_device;
        int64_t* J_device;

        T* R_device;
        T* D_device;

        CQRRPBenchData(int64_t m, int64_t n) :
        A(m * n, 0.0)
        {
            row = m;
            col = n;
            hipMalloc(&A_device,    m * n * sizeof(T));
            hipMalloc(&tau_device,  n *     sizeof(T));
            hipMalloc(&J_device,    n *     sizeof(int64_t));
            hipMalloc(&R_device,    n * n * sizeof(T));
            hipMalloc(&D_device,    n *     sizeof(T));
        }

        ~CQRRPBenchData() {
            hipFree(A_device);
            hipFree(tau_device);
            hipFree(J_device);
            hipFree(R_device);
            hipFree(D_device);
        }
    };

    template <typename T, typename RNG>
    static void data_regen(
                            RandLAPACK::gen::mat_gen_info<T> m_info, 
                            CQRRPBenchData<T> &all_data, 
                            RandBLAS::RNGState<RNG> &state) {

        auto state_const = state;
        auto m = m_info.rows;
        auto n = m_info.cols;

        RandLAPACK::gen::mat_gen(m_info, all_data.A.data(), state_const);
        hipMemset(all_data.J_device, 0.0, n);
        hipMemset(all_data.tau_device, 0.0, n);
    }

    template <typename T, typename RNG>
    static void bench_CQRRP(
        bool profile_runtime,
        bool run_qrf,
        RandLAPACK::gen::mat_gen_info<T> m_info,
        int64_t d_factor, 
        T tol,
        int64_t block_size,
        CQRRPBenchData<T> &all_data,
        RandBLAS::RNGState<RNG> state,
        std::string output_filename_breakdown,
        std::string output_filename_speed) {

        auto m = all_data.row;
        auto n = all_data.col;
        auto state_const = state;
        auto d = d_factor * block_size;

        // Skethcing in an sampling regime
        hipMalloc(&all_data.A_sk_device, d * n * sizeof(T));
        all_data.A_sk  = ( T * ) calloc( d * n, sizeof( T ) );
        T* S           = ( T * ) calloc( d * m, sizeof( T ) );
        RandBLAS::DenseDist D(d, m);
        RandBLAS::fill_dense(D, S, state_const).second;
        blas::gemm(Layout::ColMajor, Op::NoTrans, Op::NoTrans, d, n, m, 1.0, S, d, all_data.A.data(), m, 0.0, all_data.A_sk, d);
        free(S);
        hipMemcpy(all_data.A_sk_device, all_data.A_sk, d * n * sizeof(double), hipMemcpyHostToDevice);
	
        RandLAPACK::CQRRP_blocked_GPU<double, r123::Philox4x32> CQRRP_GPU(profile_runtime, tol, block_size);
        //CQRRP_GPU.use_qrf = true;
	    auto start = std::chrono::steady_clock::now();
        CQRRP_GPU.call(m, n, all_data.A_device, m, all_data.A_sk_device, d, all_data.tau_device, all_data.J_device);
	    auto stop = std::chrono::steady_clock::now();
	    auto diff = std::chrono::duration_cast<std::chrono::milliseconds>(stop - start).count();
        auto rank = CQRRP_GPU.rank;
        //printf("RANK AS RETURNED BY CQRRP GPU %4ld\n", rank);
        data_regen(m_info, all_data, state);
        hipFree(all_data.A_sk_device);
        free(all_data.A_sk);

	    printf("  BLOCK SIZE = %ld TIME (MS) = %ld\n", block_size, diff);
        std::ofstream file(output_filename_speed, std::ios::app);
        file << m << "  " << n << "  " << block_size << "  " << diff << "\n";

        if(profile_runtime) {
            std::ofstream file(output_filename_breakdown, std::ios::app);
            std::copy(CQRRP_GPU.times.data(), CQRRP_GPU.times.data() + 17, std::ostream_iterator<T>(file, ", "));
            file << "\n";
        } 
        if (run_qrf) {
            lapack::Queue lapack_queue(0);
            using lapack::device_info_int;
            device_info_int* d_info = blas::device_malloc< device_info_int >( 1, lapack_queue );
            char* d_work_geqrf;
            char* h_work_geqrf;
            size_t d_size_geqrf, h_size_geqrf;
            
            auto start_qrf = std::chrono::steady_clock::now();
            lapack::geqrf_work_size_bytes(m, n, all_data.A_device, m, &d_size_geqrf, &h_size_geqrf, lapack_queue);
            d_work_geqrf = blas::device_malloc< char >( d_size_geqrf, lapack_queue );
            std::vector<char> h_work_geqrf_vector( h_size_geqrf );
            h_work_geqrf = h_work_geqrf_vector.data();
            lapack::geqrf(m, n, all_data.A_device, m, all_data.tau_device, d_work_geqrf, d_size_geqrf, h_work_geqrf, h_size_geqrf, d_info, lapack_queue);
            lapack_queue.sync();
            auto stop_qrf  = std::chrono::steady_clock::now();
	        auto diff_qrf  = std::chrono::duration_cast<std::chrono::milliseconds>(stop_qrf  - start_qrf).count();
            printf(" QRF TIME (MS) = %ld\n", diff_qrf);
        }
    }

    template <typename T, typename RNG>
    static void bench_CholQR(
        RandLAPACK::gen::mat_gen_info<T> m_info,
        int64_t numcols,
        CQRRPBenchData<T> &all_data,
        RandBLAS::RNGState<RNG> state,
        std::string output_filename) {

        auto m = all_data.row;
        auto n = numcols;
        auto state_const = state;

        // Initialize GPU stuff
        lapack::Queue lapack_queue(0);
        hipStream_t strm = lapack_queue.stream();
        using lapack::device_info_int;
        device_info_int* d_info = blas::device_malloc< device_info_int >( 1, lapack_queue );
        char* d_work_geqrf;
        char* h_work_geqrf;
        size_t d_size_geqrf, h_size_geqrf;

        // CholQR part
        auto start_cholqr = std::chrono::steady_clock::now();
        blas::syrk(Layout::ColMajor, Uplo::Upper, Op::Trans, n, m, (T) 1.0, all_data.A_device, m, (T) 0.0, all_data.R_device, n, lapack_queue);
        lapack::potrf(Uplo::Upper,  n, all_data.R_device, n, d_info, lapack_queue);
        blas::trsm(Layout::ColMajor, Side::Right, Uplo::Upper, Op::NoTrans, Diag::NonUnit, m, n, (T) 1.0, all_data.R_device, n, all_data.A_device, m, lapack_queue);
        lapack_queue.sync();
        auto stop_cholqr  = std::chrono::steady_clock::now();
        auto diff_cholqr  = std::chrono::duration_cast<std::chrono::milliseconds>(stop_cholqr  - start_cholqr).count();
        
        auto start_orhr_col = std::chrono::steady_clock::now();
        // ORHR_COL part
        RandLAPACK::cuda_kernels::orhr_col_gpu(strm, m, n, all_data.A_device, m, all_data.tau_device, all_data.D_device);  
        RandLAPACK::cuda_kernels::R_cholqr_signs_gpu(strm, n, n, all_data.R_device, all_data.D_device);
        hipStreamSynchronize(strm);
        auto stop_orhr_col  = std::chrono::steady_clock::now();
        auto diff_orhr_col  = std::chrono::duration_cast<std::chrono::milliseconds>(stop_orhr_col  - start_orhr_col).count();

        // Mandatory data re-generation
        data_regen(m_info, all_data, state);

        // QRF part
        auto start_qrf = std::chrono::steady_clock::now();
        lapack::geqrf_work_size_bytes(m, n, all_data.A_device, m, &d_size_geqrf, &h_size_geqrf, lapack_queue);
        d_work_geqrf = blas::device_malloc< char >( d_size_geqrf, lapack_queue );
        std::vector<char> h_work_geqrf_vector( h_size_geqrf );
        h_work_geqrf = h_work_geqrf_vector.data();
        lapack::geqrf(m, n, all_data.A_device, m, all_data.tau_device, d_work_geqrf, d_size_geqrf, h_work_geqrf, h_size_geqrf, d_info, lapack_queue);
        lapack_queue.sync();
        auto stop_qrf  = std::chrono::steady_clock::now();
        auto diff_qrf  = std::chrono::duration_cast<std::chrono::milliseconds>(stop_qrf  - start_qrf).count();
        printf(" CholQR TIME (MS)   = %ld\n", diff_cholqr);
        printf(" ORHR_COL TIME (MS) = %ld\n", diff_orhr_col);
        printf(" QRF TIME (MS)      = %ld\n", diff_qrf);

        std::ofstream file(output_filename, std::ios::app);
        file << m << "  " << n << "  " << diff_cholqr << "  " << diff_orhr_col << "  " << diff_qrf << "\n";
    }

};
/*
TEST_P(BenchCQRRP, CQRRP_GPU_benchmark_16k) {
    int64_t m            = std::pow(2, 14);
    int64_t n            = std::pow(2, 14);
    double d_factor      = 1.25;
    int64_t b_sz         = GetParam();
    double tol           = std::pow(std::numeric_limits<double>::epsilon(), 0.85);
    auto state           = RandBLAS::RNGState();
    bool profile_runtime = true;
    bool run_qrf         = false;
    if(b_sz == 120) {
        run_qrf = true;
    }

    CQRRPBenchData<double> all_data(m, n);
    RandLAPACK::gen::mat_gen_info<double> m_info(m, n, RandLAPACK::gen::gaussian);
    RandLAPACK::gen::mat_gen<double, r123::Philox4x32>(m_info, all_data.A.data(), state);
    hipMemcpy(all_data.A_device, all_data.A.data(), m * n * sizeof(double), hipMemcpyHostToDevice);


    std::string file1 = "ICQRRP_GPU_runtime_breakdown_rows_"    + std::to_string(m)
                                    + "_cols_"       + std::to_string(n)
                                    + "_d_factor_"   + std::to_string(d_factor)
                                    + ".dat";

    std::string file2 = "ICQRRP_GPU_speed_rows_"    + std::to_string(m)
                                    + "_cols_"       + std::to_string(n)
                                    + "_d_factor_"   + std::to_string(d_factor)
                                    + ".dat";

    bench_CQRRP(profile_runtime, run_qrf, m_info, d_factor, tol, b_sz, all_data, state, file1, file2);
}

INSTANTIATE_TEST_SUITE_P(
    CQRRP_GPU_16k_benchmarks,
    BenchCQRRP,
    ::testing::Values(32, 40, 48, 56, 64, 72, 80, 88, 96, 104, 112, 120, 128, 136, 144, 152, 160, 168, 176, 
    184, 192, 200, 208, 216, 224, 232, 240, 248, 256, 264, 272, 280, 288, 296, 304, 312, 320, 328, 336, 344, 
    352, 360, 368, 376, 384, 392, 400, 408, 416, 424, 432, 440, 448, 456, 464, 472, 480, 488, 496, 504, 512)
);

TEST_F(BenchCQRRP, Bench_CholQR) {
    int64_t m       = std::pow(2, 14);
    int64_t n_start = 120;
    int64_t n_stop  = std::pow(2, 14);
    auto state      = RandBLAS::RNGState();

    CQRRPBenchData<double> all_data(m, n_stop);
    RandLAPACK::gen::mat_gen_info<double> m_info(m, n_stop, RandLAPACK::gen::gaussian);
    RandLAPACK::gen::mat_gen<double, r123::Philox4x32>(m_info, all_data.A.data(), state);
    hipMemcpy(all_data.A_device, all_data.A.data(), m * n_stop * sizeof(double), hipMemcpyHostToDevice);


    std::string file = "CholQR_GPU_speed_rows_"      + std::to_string(m)
                                    + "_cols_start_" + std::to_string(n_start)
                                    + "_cols_stop_"  + std::to_string(n_stop)
                                    + ".dat";

    for(int i = n_start; i <= n_stop; i += n_start)
        bench_CholQR(m_info, i, all_data, state, file);
}
*/
#endif
