#include "hip/hip_runtime.h"
#include "RandLAPACK.hh"
#include "rl_blaspp.hh"
#include "rl_lapackpp.hh"
#include "rl_gen.hh"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>

#include <RandBLAS.hh>
#include <fstream>
#include <gtest/gtest.h>
#include <chrono>
#include <numeric>

// Use cuda kernels.
#ifndef USE_CUDA
#define USE_CUDA
#include "RandLAPACK/drivers/rl_bqrrp_gpu.hh"

using GPUSubroutines = RandLAPACK::BQRRPGPUSubroutines;

class BenchBQRRP : public ::testing::Test
{
    protected:

    virtual void SetUp() {};

    virtual void TearDown() {};

    template <typename T>
    struct BQRRPBenchData {
        int64_t row;
        int64_t col;
        int64_t rank;
        
        std::vector<T> A;
        T* A_sk;
        // Buffers for the GPU data
        T* A_device;
        T* A_sk_device;
        T* tau_device;
        int64_t* J_device;

        T* R_device;
        T* D_device;

        BQRRPBenchData(int64_t m, int64_t n) :
        A(m * n, 0.0)
        {
            row = m;
            col = n;
            hipMalloc(&A_device,    m * n * sizeof(T));
            hipMalloc(&tau_device,  n *     sizeof(T));
            hipMalloc(&J_device,    n *     sizeof(int64_t));
            hipMalloc(&R_device,    n * n * sizeof(T));
            hipMalloc(&D_device,    n *     sizeof(T));
        }

        ~BQRRPBenchData() {
            hipFree(A_device);
            hipFree(tau_device);
            hipFree(J_device);
            hipFree(R_device);
            hipFree(D_device);
        }
    };

    template <typename T, typename RNG>
    static void data_regen(
                            RandLAPACK::gen::mat_gen_info<T> m_info, 
                            BQRRPBenchData<T> &all_data, 
                            RandBLAS::RNGState<RNG> &state) {

        auto state_const = state;
        auto m = m_info.rows;
        auto n = m_info.cols;

        RandLAPACK::gen::mat_gen(m_info, all_data.A.data(), state_const);
        hipMemset(all_data.J_device, 0.0, n);
        hipMemset(all_data.tau_device, 0.0, n);
    }

    template <typename T, typename RNG>
    static void bench_BQRRP(
        bool profile_runtime,
        bool run_qrf,
        RandLAPACK::gen::mat_gen_info<T> m_info,
        int64_t m,
        int64_t n,
        int64_t block_size,
        BQRRPBenchData<T> &all_data,
        RandBLAS::RNGState<RNG> state,
        std::string* output_filename_breakdown_QRF,
        std::string* output_filename_breakdown_CholQR,
        std::string* output_filename_speed) {

	    T d_factor = 1.0;
        auto state_const = state;
        int64_t d = d_factor * block_size;

        // BQRRP with QRF
        // Skethcing in an sampling regime
        hipMalloc(&all_data.A_sk_device, d * n * sizeof(T));
        all_data.A_sk = new T[d * n]();
        T* S          = new T[d * m]();

        RandBLAS::DenseDist D(d, m);
        RandBLAS::fill_dense(D, S, state_const);
        blas::gemm(Layout::ColMajor, Op::NoTrans, Op::NoTrans, d, n, m, 1.0, S, d, all_data.A.data(), m, 0.0, all_data.A_sk, d);
        hipMemcpy(all_data.A_sk_device, all_data.A_sk, d * n * sizeof(double), hipMemcpyHostToDevice);
        RandLAPACK::BQRRP_GPU<double, r123::Philox4x32> BQRRP_GPU_QRF(profile_runtime, block_size);
        BQRRP_GPU_QRF.qr_tall = GPUSubroutines::QRTall::geqrf;
	    auto start_bqrrp_qrf = std::chrono::steady_clock::now();
        BQRRP_GPU_QRF.call(m, n, all_data.A_device, m, all_data.A_sk_device, d, all_data.tau_device, all_data.J_device);
        auto stop_bqrrp_qrf = std::chrono::steady_clock::now();
	    auto diff_bqrrp_qrf = std::chrono::duration_cast<std::chrono::microseconds>(stop_bqrrp_qrf - start_bqrrp_qrf).count();
        data_regen(m_info, all_data, state);
        hipFree(all_data.A_sk_device);
        delete[] all_data.A_sk;

        if(profile_runtime) {
            std::ofstream file(*output_filename_breakdown_QRF, std::ios::app);
            std::copy(BQRRP_GPU_QRF.times.data(), BQRRP_GPU_QRF.times.data() + 15, std::ostream_iterator<T>(file, ", "));
            file << "\n";
        } 

        // BQRRP with CholQR
        // Skethcing in an sampling regime
        hipMalloc(&all_data.A_sk_device, d * n * sizeof(T));
        all_data.A_sk = new T[d * n]();
        blas::gemm(Layout::ColMajor, Op::NoTrans, Op::NoTrans, d, n, m, 1.0, S, d, all_data.A.data(), m, 0.0, all_data.A_sk, d);
        delete[] S;
        hipMemcpy(all_data.A_sk_device, all_data.A_sk, d * n * sizeof(double), hipMemcpyHostToDevice);
        RandLAPACK::BQRRP_GPU<double, r123::Philox4x32> BQRRP_GPU_CholQR(profile_runtime, block_size);
        BQRRP_GPU_CholQR.qr_tall = GPUSubroutines::QRTall::cholqr;
	    auto start_bqrrp_cholqr = std::chrono::steady_clock::now();
        BQRRP_GPU_CholQR.call(m, n, all_data.A_device, m, all_data.A_sk_device, d, all_data.tau_device, all_data.J_device);
	    auto stop_bqrrp_cholqr = std::chrono::steady_clock::now();
	    auto diff_bqrrp_cholqr = std::chrono::duration_cast<std::chrono::microseconds>(stop_bqrrp_cholqr - start_bqrrp_cholqr).count();
        data_regen(m_info, all_data, state);
        hipFree(all_data.A_sk_device);
        delete[] all_data.A_sk;

        if(profile_runtime) {
            std::ofstream file(*output_filename_breakdown_CholQR, std::ios::app);
            std::copy(BQRRP_GPU_CholQR.times.data(), BQRRP_GPU_CholQR.times.data() + 15, std::ostream_iterator<T>(file, ", "));
            file << "\n";
        } 

        // Optional QRF
        long diff_qrf = 0;
        if (run_qrf) {
            lapack::Queue lapack_queue(0);
            using lapack::device_info_int;
            device_info_int* d_info = blas::device_malloc< device_info_int >( 1, lapack_queue );
            char* d_work_geqrf;
            char* h_work_geqrf;
            size_t d_size_geqrf, h_size_geqrf;
            
            auto start_qrf = std::chrono::steady_clock::now();
            lapack::geqrf_work_size_bytes(m, n, all_data.A_device, m, &d_size_geqrf, &h_size_geqrf, lapack_queue);
            d_work_geqrf = blas::device_malloc< char >( d_size_geqrf, lapack_queue );
            std::vector<char> h_work_geqrf_vector( h_size_geqrf );
            h_work_geqrf = h_work_geqrf_vector.data();
            lapack::geqrf(m, n, all_data.A_device, m, all_data.tau_device, d_work_geqrf, d_size_geqrf, h_work_geqrf, h_size_geqrf, d_info, lapack_queue);
            lapack_queue.sync();
            auto stop_qrf  = std::chrono::steady_clock::now();
	        diff_qrf  = std::chrono::duration_cast<std::chrono::microseconds>(stop_qrf  - start_qrf).count();
            printf(" QRF TIME (MS) = %ld\n", diff_qrf);
        }

	    printf("  BLOCK SIZE = %ld BQRRP+QRF TIME (MS) = %ld BQRRP+CholQR TIME (MS) = %ld\n", block_size, diff_bqrrp_qrf, diff_bqrrp_cholqr);
        std::ofstream file(*output_filename_speed, std::ios::app);
        file << diff_bqrrp_qrf << "  " << diff_bqrrp_cholqr << "  " << diff_qrf << "\n";
        hipError_t ierr = hipGetLastError();
    	if (ierr != hipSuccess)
    	{
        	RandLAPACK_CUDA_ERROR("Error before bench_bqrrp returned. " << hipGetErrorString(ierr))
        	abort();
    	}
    }

    // Not using this right now. But there's no harm in keeping it around.
    template <typename T, typename RNG>
    static void bench_CholQR(
        RandLAPACK::gen::mat_gen_info<T> m_info,
        int64_t numrows,
        BQRRPBenchData<T> &all_data,
        RandBLAS::RNGState<RNG> state,
        std::string output_filename) {

        auto m = numrows;
        auto n = all_data.col;
        auto state_const = state;

        // Initialize GPU stuff
        lapack::Queue lapack_queue(0);
        hipStream_t strm = lapack_queue.stream();
        using lapack::device_info_int;
        device_info_int* d_info = blas::device_malloc< device_info_int >( 1, lapack_queue );
        char* d_work_geqrf;
        char* h_work_geqrf;
        size_t d_size_geqrf, h_size_geqrf;

        // CholQR part
        auto start_cholqr = std::chrono::steady_clock::now();
        blas::syrk(Layout::ColMajor, Uplo::Upper, Op::Trans, n, m, (T) 1.0, all_data.A_device, m, (T) 0.0, all_data.R_device, n, lapack_queue);
        lapack::potrf(Uplo::Upper,  n, all_data.R_device, n, d_info, lapack_queue);
        blas::trsm(Layout::ColMajor, Side::Right, Uplo::Upper, Op::NoTrans, Diag::NonUnit, m, n, (T) 1.0, all_data.R_device, n, all_data.A_device, m, lapack_queue);
        lapack_queue.sync();
        auto stop_cholqr  = std::chrono::steady_clock::now();
        auto diff_cholqr  = std::chrono::duration_cast<std::chrono::microseconds>(stop_cholqr  - start_cholqr).count();
        
        auto start_orhr_col = std::chrono::steady_clock::now();
        // ORHR_COL part
        RandLAPACK::cuda_kernels::orhr_col_gpu(strm, m, n, all_data.A_device, m, all_data.tau_device, all_data.D_device);  
        RandLAPACK::cuda_kernels::R_cholqr_signs_gpu(strm, n, n, all_data.R_device, all_data.D_device);
        hipStreamSynchronize(strm);
        auto stop_orhr_col  = std::chrono::steady_clock::now();
        auto diff_orhr_col  = std::chrono::duration_cast<std::chrono::microseconds>(stop_orhr_col  - start_orhr_col).count();

        // Mandatory data re-generation
        data_regen(m_info, all_data, state);

        // QRF part
        auto start_qrf = std::chrono::steady_clock::now();
        lapack::geqrf_work_size_bytes(m, n, all_data.A_device, m, &d_size_geqrf, &h_size_geqrf, lapack_queue);
        d_work_geqrf = blas::device_malloc< char >( d_size_geqrf, lapack_queue );
        std::vector<char> h_work_geqrf_vector( h_size_geqrf );
        h_work_geqrf = h_work_geqrf_vector.data();
        lapack::geqrf(m, n, all_data.A_device, m, all_data.tau_device, d_work_geqrf, d_size_geqrf, h_work_geqrf, h_size_geqrf, d_info, lapack_queue);
        lapack_queue.sync();
        auto stop_qrf  = std::chrono::steady_clock::now();
        auto diff_qrf  = std::chrono::duration_cast<std::chrono::microseconds>(stop_qrf  - start_qrf).count();
        printf(" CholQR TIME (MS)   = %ld\n", diff_cholqr);
        printf(" ORHR_COL TIME (MS) = %ld\n", diff_orhr_col);
        printf(" QRF TIME (MS)      = %ld\n", diff_qrf);

        std::ofstream file(output_filename, std::ios::app);
        file << diff_cholqr << "  " << diff_orhr_col << "  " << diff_qrf << "\n";

        hipError_t ierr = hipGetLastError();
    	if (ierr != hipSuccess)
    	{
        	RandLAPACK_CUDA_ERROR("Error before bench_CholQR returned. " << hipGetErrorString(ierr))
        	abort();
    	}    
    }

    static void setup_bqrrp_speed_comparisons_block_size(
        int64_t m,
        int64_t n,
        std::vector<int64_t> b_sz 
    ){
        // Get a string representation of the block size vector
        std::string b_sz_string = std::accumulate(b_sz.begin(), b_sz.end(), std::string(), 
                                    [](const std::string& a, int b) {
                                        return a.empty() ? std::to_string(b) : a + "," + std::to_string(b);
                                    });

        auto state           = RandBLAS::RNGState();
        bool profile_runtime = true;
        bool run_qrf         = true;

        BQRRPBenchData<double> all_data(m, n);
        RandLAPACK::gen::mat_gen_info<double> m_info(m, n, RandLAPACK::gen::gaussian);
        RandLAPACK::gen::mat_gen<double, r123::Philox4x32>(m_info, all_data.A.data(), state);
        hipMemcpy(all_data.A_device, all_data.A.data(), m * n * sizeof(double), hipMemcpyHostToDevice);

        std::string* file_name_1 = new std::string(RandLAPACK::util::getCurrentDate<double>() + "BQRRP_GPU_runtime_breakdown_qrf_"       
                            + "_num_info_lines_" + std::to_string(7) +
                            ".txt");

        std::string* file_name_2 = new std::string(RandLAPACK::util::getCurrentDate<double>() + "BQRRP_GPU_runtime_breakdown_cholqr_"  
                            + "_num_info_lines_" + std::to_string(7) +
                              ".txt");

        std::string* file_name_3 = new std::string(RandLAPACK::util::getCurrentDate<double>() + "BQRRP_GPU_speed_comparisons_block_size"  
                            + "_num_info_lines_" + std::to_string(7) +
                              ".txt");

        std::ofstream file1(*file_name_1, std::ios::out | std::ios::app);
        std::ofstream file2(*file_name_2, std::ios::out | std::ios::app);
        std::ofstream file3(*file_name_3, std::ios::out | std::ios::app);

        file1 << "Description: Results from the BQRRP GPU runtime breakdown benchmark, recording the time it takes to perform every subroutine in BQRRP."
                "\nFile format: 15 data columns, each corresponding to a given BQRRP subroutine: preallocation_t_dur, qrcp_main_t_dur, copy_A_sk_t_dur, qrcp_piv_t_dur, copy_A_t_dur, piv_A_t_dur, copy_J_t_dur, updating_J_t_dur, preconditioning_t_dur, qr_tall_t_dur, q_reconstruction_t_dur, apply_transq_t_dur, sample_update_t_dur, t_rest, total_t_dur"
                "               rows correspond to BQRRP runs with block sizes varying in a way unique for a particular run."
                "\nNum OMP threads:"  + std::to_string(RandLAPACK::util::get_omp_threads()) +
                "\nInput type:"       + std::to_string(m_info.m_type) +
                "\nInput size:"       + std::to_string(m) + " by "  + std::to_string(n) +
                "\nAdditional parameters: Tall QR subroutine cholqr BQRRP block sizes: " + b_sz_string +
                "\n";
        file1.flush();

        file2 << "Description: Results from the BQRRP GPU runtime breakdown benchmark, recording the time it takes to perform every subroutine in BQRRP."
                "\nFile format: 15 data columns, each corresponding to a given BQRRP subroutine: preallocation_t_dur, qrcp_main_t_dur, copy_A_sk_t_dur, qrcp_piv_t_dur, copy_A_t_dur, piv_A_t_dur, copy_J_t_dur, updating_J_t_dur, preconditioning_t_dur, qr_tall_t_dur, q_reconstruction_t_dur, apply_transq_t_dur, sample_update_t_dur, t_rest, total_t_dur"
                "               rows correspond to BQRRP runs with block sizes varying in a way unique for a particular run."
                "\nNum OMP threads:"  + std::to_string(RandLAPACK::util::get_omp_threads()) +
                "\nInput type:"       + std::to_string(m_info.m_type) +
                "\nInput size:"       + std::to_string(m) + " by "  + std::to_string(n) +
                "\nAdditional parameters: Tall QR subroutine geqrf BQRRP block sizes: " + b_sz_string +
                "\n";
        file2.flush();

        file3 << "Description: Results from the BQRRP GPU speed comparison benchmark, recording the time it takes to perform BQRRP and alternative QR and QRCP factorizations."
                "\nFile format: 3 columns, containing time for each algorithm: BQRRP+CholQR, BQRRP+QRF, QRF;"
                "               rows correspond to BQRRP runs with block sizes varying in powers of 2 or multiples of 10"
                "\nNum OMP threads:"  + std::to_string(RandLAPACK::util::get_omp_threads()) +
                "\nInput type:"       + std::to_string(m_info.m_type) +
                "\nInput size:"       + std::to_string(m) + " by "  + std::to_string(n) +
                "\nAdditional parameters: BQRRP block sizes: " + b_sz_string +
                "\n";
        file3.flush();

        for(size_t i = 0; i < b_sz.size(); ++i) {
            bench_BQRRP(profile_runtime, run_qrf, m_info, m, n, b_sz[i], all_data, state, file_name_1, file_name_2, file_name_3);
            run_qrf = false;
        }
    }

    static void setup_bqrrp_speed_comparisons_mat_size(
        std::vector<int64_t> m_sz,
        int64_t b_sz 
    ){
        // Get a string representation of the block size vector
        std::string m_sz_string = std::accumulate(m_sz.begin(), m_sz.end(), std::string(), 
                                    [](const std::string& a, int b) {
                                        return a.empty() ? std::to_string(b) : a + "," + std::to_string(b);
                                    });

        auto state           = RandBLAS::RNGState();
        bool profile_runtime = false;
        bool run_qrf         = true;

        int64_t m_max = *std::max_element(m_sz.begin(), m_sz.end());
        BQRRPBenchData<double> all_data(m_max, m_max);
        RandLAPACK::gen::mat_gen_info<double> m_info(m_max, m_max, RandLAPACK::gen::gaussian);
        RandLAPACK::gen::mat_gen<double, r123::Philox4x32>(m_info, all_data.A.data(), state);
        hipMemcpy(all_data.A_device, all_data.A.data(), m_max * m_max * sizeof(double), hipMemcpyHostToDevice);

        std::string* file_name = new std::string(RandLAPACK::util::getCurrentDate<double>() + "BQRRP_GPU_speed_comparisons_mat_size"  
                            + "_num_info_lines_" + std::to_string(7) +
                              ".txt");

        std::ofstream file(*file_name, std::ios::out | std::ios::app);
        file << "Description: Results from the BQRRP GPU speed comparison benchmark, recording the time it takes to perform BQRRP and alternative QR and QRCP factorizations."
                "\nFile format: 7 columns, containing time for each algorithm: BQRRP+CholQR, BQRRP+QRF, QRF;"
                "               rows correspond to BQRRP runs with varying mat sizes, with numruns repititions of each mat size."
                "\nNum OMP threads:"  + std::to_string(RandLAPACK::util::get_omp_threads()) +
                "\nInput type:"       + std::to_string(m_info.m_type) +
                "\nInput size:"       + " dim start: " + m_sz_string +
                "\nAdditional parameters: BQRRP block size: " + std::to_string(b_sz) + 
                "\n";
        file.flush();

        for(size_t i = 0; i < m_sz.size(); ++i) {
            bench_BQRRP(profile_runtime, run_qrf, m_info, m_sz[i], m_sz[i], m_sz[i]/32, all_data, state, nullptr, nullptr, file_name);
            run_qrf = false;
        }
    }
};

TEST_F(BenchBQRRP, BQRRP_GPU_block_sizes_powers_of_two) {
    int64_t m                 = std::pow(2, 15);
    int64_t n                 = std::pow(2, 15);
    std::vector<int64_t> b_sz = {32, 64, 96, 128, 160, 192, 224, 256, 288, 320, 352, 384, 416, 448, 480, 512, 640, 768, 896, 1024, 1152, 1280, 1408, 1536, 1664, 1792, 1920, 2048};
    setup_bqrrp_speed_comparisons_block_size(m, n, b_sz);
}

TEST_F(BenchBQRRP, BQRRP_GPU_block_sizes_multiples_of_ten) {
    int64_t m                 = 32000;
    int64_t n                 = 32000;
    std::vector<int64_t> b_sz = {50, 100, 150, 200, 250, 300, 350, 400, 450, 500, 550, 600, 650, 700, 750, 800, 850, 900, 950, 1000, 1000, 1125, 1250, 1375, 1500, 1625, 1750, 1875, 2000};
    setup_bqrrp_speed_comparisons_block_size(m, n, b_sz);
}

TEST_F(BenchBQRRP, BQRRP_GPU_mat_sizes_powers_of_two) {
    std::vector<int64_t> m_sz = {512, 1024, 2048, 4096, 8192, 32768};
    int64_t b_sz              = 0;
    setup_bqrrp_speed_comparisons_mat_size(m_sz, b_sz);
}

TEST_F(BenchBQRRP, BQRRP_GPU_mat_sizes_multiples_of_ten) {
    std::vector<int64_t> m_sz = {512, 1000, 2000, 4000, 8000, 32000};
    int64_t b_sz              = 0;
    setup_bqrrp_speed_comparisons_mat_size(m_sz, b_sz);
}
#endif
